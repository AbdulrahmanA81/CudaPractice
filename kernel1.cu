#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2022 Bogdan Simion
 * -------------
 */

#include "kernels.h"
#include <iostream>

#define MAX_FILTER_DIM 9 

TimingData run_kernel1(const int8_t *filter, int32_t dimension, const int32_t *input,
                 int32_t *output, int32_t width, int32_t height) {
    // Figure out how to split the work into threads and call the kernel below.

    // Allocate GPU memory
    int32_t *d_input, *d_output;
    int8_t *d_filter;
    TimingData timing_data;

    hipMalloc((void **)&d_input, width * height * sizeof(int32_t));
    hipMalloc((void **)&d_output, width * height * sizeof(int32_t));
    hipMalloc((void **)&d_filter, dimension * dimension * sizeof(int8_t));

    Clock clock;
    // Copy data into GPU
    clock.start();
    hipMemcpy(d_input, input, width * height * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, dimension * dimension * sizeof(int8_t), hipMemcpyHostToDevice);

    timing_data.time_gpu_transfer_in = clock.stop();

    // Initializing block size
    dim3 blockSize(16, 16); 
    dim3 gridSize((height + blockSize.x - 1) / blockSize.x, (width + blockSize.y - 1) / blockSize.y);


    clock.start();
    kernel1<<<gridSize, blockSize>>>(d_filter, dimension, d_input, d_output, width, height);
    hipDeviceSynchronize();  // Ensure kernel execution completes
    timing_data.time_gpu_computation = clock.stop();

    int32_t *d_mins, *d_maxes;

    // Reduction for min/max
    int blocksPerColumn = (width + blockSize.x - 1) / blockSize.x;
    int blocksPerRow = height; // One block for each row
    dim3 gridDimReduction(blocksPerColumn, blocksPerRow);

    int totalBlocks = blocksPerColumn * blocksPerRow;
    hipMalloc((void **)&d_mins, totalBlocks * sizeof(int32_t));
    hipMalloc((void **)&d_maxes, totalBlocks * sizeof(int32_t));

    // Launch findMinMax kernel with 2D grid
    clock.start();
    findMinMax<<<gridDimReduction, blockSize, 2 * blockSize.x * sizeof(int32_t)>>>(d_output, d_mins, d_maxes, width, height);
    hipDeviceSynchronize();
    timing_data.time_gpu_computation = timing_data.time_gpu_computation + clock.stop();

    // Copy min/max arrays back to host
    int32_t *mins = new int32_t[totalBlocks];
    int32_t *maxes = new int32_t[totalBlocks];
    hipMemcpy(mins, d_mins, totalBlocks * sizeof(int32_t), hipMemcpyDeviceToHost);
    hipMemcpy(maxes, d_maxes, totalBlocks * sizeof(int32_t), hipMemcpyDeviceToHost);

    // CPU reduction to find global min/max
    int32_t globalMin = INT_MAX;
    int32_t globalMax = INT_MIN;
    for (int i = 0; i < totalBlocks; i++) {
        globalMin = min(globalMin, mins[i]);
        globalMax = max(globalMax, maxes[i]);
    }

    clock.start();
    normalize1<<<gridSize, blockSize>>>(d_output, width, height, globalMin, globalMax);
    hipDeviceSynchronize();
    timing_data.time_gpu_computation = timing_data.time_gpu_computation + clock.stop();


    // Copy the result back out
    clock.start();
    hipMemcpy(output, d_output, width * height * sizeof(int32_t), hipMemcpyDeviceToHost);
    timing_data.time_gpu_transfer_out = clock.stop();
    // free allocated memory

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_filter);

    return timing_data;

}

__global__ void kernel1(const int8_t *filter, int32_t dimension,
                        const int32_t *input, int32_t *output, int32_t width,
                        int32_t height) {


    __shared__ int8_t sharedFilter[MAX_FILTER_DIM][MAX_FILTER_DIM];


    // Load the filter into shared memory
    // Each thread loads one element of the filter
    int filterX = threadIdx.x;
    int filterY = threadIdx.y;

    if (filterX < dimension && filterY < dimension) {
      sharedFilter[filterY][filterX] = filter[filterY * dimension + filterX];
    }

    // Synchronize to make sure the filter is fully loaded
    __syncthreads();


    // In a column-major approach, the 'col' index is determined by the block index and thread index in Y direction.
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    // The 'row' index is determined by the block index and thread index in X direction.
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread is within the image boundaries
    if (col < width && row < height) {
        int result = 0;
        int center = dimension / 2;

        // Apply the filter centered around the current pixel
        for (int i = -center; i <= center; i++) {
            for (int j = -center; j <= center; j++) {
                int curRow = row + i;
                int curCol = col + j;

                // Check if the neighboring pixel is within the image boundaries
                if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width) {
                    int imgIdx = curRow * width + curCol;
                    
                    result += input[imgIdx] *  sharedFilter[i + center][j + center];
                }
            }
        }
        // Store the result in the output image
        output[row * width + col] = result;
    }    
}

__global__ void findMinMax(const int32_t *input, int32_t *mins, int32_t *maxes, int32_t width, int32_t height) {
    extern __shared__ int32_t sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y;

    unsigned int index = y + x * height; // Convert to column-major index

    // Load input into shared memory.
    sdata[tid] = (index < width * height) ? input[index] : INT_MAX;
    sdata[blockDim.x + tid] = (index < width * height) ? input[index] : INT_MIN;
    __syncthreads();

    // Perform reduction in shared memory.
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = min(sdata[tid], sdata[tid + s]);
            sdata[blockDim.x + tid] = max(sdata[blockDim.x + tid], sdata[blockDim.x + tid + s]);
        }
        __syncthreads();
    }

    // Write result for this block to global memory.
    if (tid == 0) {
        mins[blockIdx.x + blockIdx.y * gridDim.x] = sdata[0]; // Index adjusted for column-major
        maxes[blockIdx.x + blockIdx.y * gridDim.x] = sdata[blockDim.x]; // Index adjusted for column-major
    }
}

__global__ void normalize1(int32_t *image, int32_t width, int32_t height, int32_t smallest, int32_t biggest) {
    // Return if there is no range to normalize.
    if (smallest == biggest) {
        return;
    }

    // In a column-major approach, the 'col' index is determined by the block index and thread index in Y direction.
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    // The 'row' index is determined by the block index and thread index in X direction.
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    int index = row * width + col;

    
    if (col < width && row < height) {
        // Perform normalization.
        image[index] = ((image[index] - smallest) * 255) / (biggest - smallest);
    }
   
}
